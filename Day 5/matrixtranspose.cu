#include<stdio.h>
#include<hip/hip_runtime.h>
#include<iostream>

#define WIDTH 1024
#define HEIGHT 1024

__global__ void transposeMatrix(const float* d_in, float* d_out, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x<width && y<height)
    {
        int inputIdx = y*width + x;
        int outputIdx = x*height + y;
        d_out[outputIdx] = d_in[inputIdx];
    }
}

void checkCudaError(const char* message) {
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << message << " - CUDA Error: " << hipGetErrorString(error) << std::endl;
        exit(EXIT_FAILURE);
    }
}

int main() {
    int width = WIDTH;
    int height = HEIGHT;

    // Allocate host memory
    size_t size = width * height * sizeof(float);
    float* h_input = (float*)malloc(size);
    float* h_output = (float*)malloc(size);

    // Initialize the input matrix with some values
    for (int i = 0; i < width * height; i++) {
        h_input[i] = static_cast<float>(i);
    }

    // Allocate device memory
    float* d_input;
    float* d_output;
    hipMalloc((void**)&d_input, size);
    hipMalloc((void**)&d_output, size);

    // Copy data from host to device
    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);
    checkCudaError("Failed to copy input data to device");

    // Define block and grid sizes
    dim3 blockSize(32, 32);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    // Launch the kernel
    transposeMatrix<<<gridSize, blockSize>>>(d_input, d_output, width, height);
    hipDeviceSynchronize();
    checkCudaError("Kernel execution failed");

    // Copy the result back to the host
    hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost);
    checkCudaError("Failed to copy output data to host");

    // Verify the result
    bool success = true;
    for (int i = 0; i < width; i++) {
        for (int j = 0; j < height; j++) {
            if (h_output[i * height + j] != h_input[j * width + i]) {
                success = false;
                break;
            }
        }
    }

    std::cout << (success ? "Matrix transposition succeeded!" : "Matrix transposition failed!") << std::endl;

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);

    // Free host memory
    free(h_input);
    free(h_output);

    return 0;
}